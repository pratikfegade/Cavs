#include "cavs/backend/op_impl.h"
#include "cavs/backend/cuda_common.h"
#include "cavs/midend/allocator.h"
#include "cavs/proto/tensor_shape.pb.h"
#include "cavs/util/macros_gpu.h"
#include "cavs/util/cudnn_types.h"

namespace backend {

using ::midend::Allocator;
using ::midend::GetAllocator;
/*using ::midend::DeviceTypeToString;*/
using ::midend::Tensor;

class ConvOpCudnnBase : public OpImpl {
 public:
  explicit ConvOpCudnnBase(const OpDef& def);
  ~ConvOpCudnnBase();

 protected:
  hipdnnTensorDescriptor_t x_desc_, y_desc_;
  hipdnnTensorDescriptor_t bias_desc_;
  hipdnnFilterDescriptor_t filter_desc_;
  hipdnnConvolutionDescriptor_t conv_desc_;
  hipdnnConvolutionFwdAlgo_t fwd_algo_;
  hipdnnConvolutionBwdFilterAlgo_t bwd_f_algo_;
  hipdnnConvolutionBwdDataAlgo_t bwd_d_algo_;
  Allocator* alloc_;
};

ConvOpCudnnBase::ConvOpCudnnBase(const OpDef& def)
    : OpImpl(def) {
  checkCUDNNError(hipdnnCreateTensorDescriptor(&x_desc_));
  checkCUDNNError(hipdnnCreateTensorDescriptor(&y_desc_));
  checkCUDNNError(hipdnnCreateTensorDescriptor(&bias_desc_));
  checkCUDNNError(hipdnnCreateFilterDescriptor(&filter_desc_));
  checkCUDNNError(hipdnnCreateConvolutionDescriptor(&conv_desc_));
  alloc_ = GetAllocator(DeviceTypeToString(GPU));
}

ConvOpCudnnBase::~ConvOpCudnnBase() {
  checkCUDNNError(hipdnnDestroyTensorDescriptor(x_desc_));
  checkCUDNNError(hipdnnDestroyTensorDescriptor(y_desc_));
  checkCUDNNError(hipdnnDestroyTensorDescriptor(bias_desc_));
  checkCUDNNError(hipdnnDestroyFilterDescriptor(filter_desc_));
  checkCUDNNError(hipdnnDestroyConvolutionDescriptor(conv_desc_));
}

template <typename T>
class ConvOpCudnn: public ConvOpCudnnBase {
 public:
  explicit ConvOpCudnn(const OpDef& def)
      : ConvOpCudnnBase(def),
        workspace(NULL), workspaceSizeInBytes(0) {}
  ~ConvOpCudnn();
  void Compute(OpContext* context) override;

 private:
  size_t workspaceSizeInBytes;
  void* workspace;
};

template <typename T>
ConvOpCudnn<T>::~ConvOpCudnn() {
  if (workspace)
    alloc_->Deallocate<char>((char*)workspace);
}

template <typename T>
void ConvOpCudnn<T>::Compute(OpContext* context) {
  const Tensor& x = context->Input(0);
  const Tensor& filter = context->Input(1);
  const Tensor& bias   = context->Input(2);
  Tensor* y = context->Output(0);

  int XN = x.dims(0);
  int XC = x.dims(1);
  int XH = x.dims(2);
  int XW = x.dims(3);
  int FYC = filter.dims(0);
  int FXC = filter.dims(1);
  int FH = filter.dims(2);
  int FW = filter.dims(3);
  int YN = y->dims(0);
  int YC = y->dims(1);
  int YH = y->dims(2);
  int YW = y->dims(3);
  CHECK(FXC == XC);
  CHECK(FYC == YC);
  CHECK(YN == XN);

  checkCUDNNError(hipdnnSetTensor4dDescriptor(bias_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  1, FYC, 1, 1));
  checkCUDNNError(hipdnnSetTensor4dDescriptor(x_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  XN, XC, XH, XW));
  checkCUDNNError(hipdnnSetTensor4dDescriptor(y_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  YN, YC, YH, YW));
  checkCUDNNError(hipdnnSetFilter4dDescriptor(filter_desc_,
                  DataTypeToCudnnType<T>::value, HIPDNN_TENSOR_NCHW,
                  FYC, FXC, FH, FW));
  checkCUDNNError(hipdnnSetConvolution2dDescriptor(conv_desc_,
                  0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION,
                  DataTypeToCudnnType<T>::value));
  /*checkCUDNNError(cudnnGetConvolutionNdForwardOutputDim(*/
                  /*conv_desc_, x_desc_, filter_desc_, */
                  /*4, YDim));*/

  // checkCUDNNError(hipdnnGetConvolutionForwardAlgorithm(CudaCommon::cudnnHandle(),
  //                 x_desc_, filter_desc_, conv_desc_, y_desc_,
  //                 HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &fwd_algo_));
  checkCUDNNError(hipdnnGetConvolutionForwardWorkspaceSize(CudaCommon::cudnnHandle(),
                  x_desc_, filter_desc_, conv_desc_, y_desc_,
                  fwd_algo_, &workspaceSizeInBytes));
  /*checkCudaError(hipMalloc((void**)&workspace, workspaceSizeInBytes));*/
  if (workspace)
    alloc_->Deallocate<char>((char*)workspace);
  workspace = alloc_->Allocate<char>(workspaceSizeInBytes);

  float alpha = 1.f, beta = 0.f;
  checkCUDNNError(hipdnnConvolutionForward(CudaCommon::cudnnHandle(),
                  &alpha, x_desc_,
                  x.data<T>(), filter_desc_, filter.data<T>(),
                  conv_desc_, fwd_algo_, workspace, workspaceSizeInBytes, &beta,
                  y_desc_, y->mutable_data<T>()));
  checkCUDNNError(hipdnnAddTensor(CudaCommon::cudnnHandle(),
                  &alpha, bias_desc_,
                  bias.data<T>(), &alpha,
                  y_desc_, y->mutable_data<T>()));

  /*x.DebugNumerical<T>();*/
  /*filter.DebugNumerical<T>();*/
  /*bias.DebugNumerical<T>();*/
  /*y->DebugNumerical<T>();*/
}

template <typename T>
class ConvOpCudnnGrad: public ConvOpCudnnBase {
 public:
  explicit ConvOpCudnnGrad(const OpDef& def)
      : ConvOpCudnnBase(def),
      filter_workspace(NULL), data_workspace(NULL),
      filter_workspaceSizeInBytes_(0), data_workspaceSizeInBytes_(0) {}
  ~ConvOpCudnnGrad();
  void Compute(OpContext* context) override;

 private:
  size_t filter_workspaceSizeInBytes_;
  size_t data_workspaceSizeInBytes_;
  void* filter_workspace;
  void* data_workspace;
};

template <typename T>
ConvOpCudnnGrad<T>::~ConvOpCudnnGrad() {
  alloc_->Deallocate<char>((char*)filter_workspace);
  alloc_->Deallocate<char>((char*)data_workspace);
}

template <typename T>
void ConvOpCudnnGrad<T>::Compute(OpContext* context) {
  const Tensor& dy = context->Input(0);
  const Tensor& x = context->Input(1);
  const Tensor& filter = context->Input(2);
  /*const Tensor& bias   = context->Input(3);*/
  Tensor* df = context->Output(0);
  Tensor* db = context->Output(1);
  Tensor* dx = context->Output(2);

  int XN = x.dims(0);
  int XC = x.dims(1);
  int XH = x.dims(2);
  int XW = x.dims(3);
  int FYC = filter.dims(0);
  int FXC = filter.dims(1);
  int FH = filter.dims(2);
  int FW = filter.dims(3);
  int YN = dy.dims(0);
  int YC = dy.dims(1);
  int YH = dy.dims(2);
  int YW = dy.dims(3);
  CHECK(FXC == XC);
  CHECK(FYC == YC);
  CHECK(YN == XN);

  checkCUDNNError(hipdnnSetTensor4dDescriptor(bias_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  1, FYC, 1, 1));
  checkCUDNNError(hipdnnSetTensor4dDescriptor(x_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  XN, XC, XH, XW));
  checkCUDNNError(hipdnnSetTensor4dDescriptor(y_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  YN, YC, YH, YW));
  checkCUDNNError(hipdnnSetFilter4dDescriptor(filter_desc_,
                  DataTypeToCudnnType<T>::value, HIPDNN_TENSOR_NCHW,
                  FYC, FXC, FH, FW));
  checkCUDNNError(hipdnnSetConvolution2dDescriptor(conv_desc_,
                  0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION,
                  DataTypeToCudnnType<T>::value));
  {
    size_t filter_worksize = 0;
    size_t data_worksize = 0;
    // checkCUDNNError(hipdnnGetConvolutionBackwardFilterAlgorithm(CudaCommon::cudnnHandle(),
    //                 x_desc_, y_desc_, conv_desc_, filter_desc_,
    //                 HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &bwd_f_algo_));
    checkCUDNNError(hipdnnGetConvolutionBackwardFilterWorkspaceSize(CudaCommon::cudnnHandle(),
                    x_desc_, y_desc_, conv_desc_, filter_desc_,
                    bwd_f_algo_, &filter_worksize));
    // checkCUDNNError(hipdnnGetConvolutionBackwardDataAlgorithm(CudaCommon::cudnnHandle(),
    //                 filter_desc_, y_desc_, conv_desc_, x_desc_,
    //                 HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &bwd_d_algo_));
    checkCUDNNError(hipdnnGetConvolutionBackwardDataWorkspaceSize(CudaCommon::cudnnHandle(),
                    filter_desc_, y_desc_, conv_desc_, x_desc_,
                    bwd_d_algo_, &data_worksize));
    VLOG(V_DEBUG) << "allocating workspace";
    if (filter_worksize != filter_workspaceSizeInBytes_) {
      filter_workspaceSizeInBytes_ = filter_worksize;
      if (filter_workspace)
        alloc_->Deallocate<char>((char*)filter_workspace);
      filter_workspace = alloc_->Allocate<char>(filter_workspaceSizeInBytes_);
    }
    if (data_worksize != data_workspaceSizeInBytes_) {
      data_workspaceSizeInBytes_ = data_worksize;
      if (data_workspace)
        alloc_->Deallocate<char>((char*)data_workspace);
      data_workspace = alloc_->Allocate<char>(data_workspaceSizeInBytes_);
    }
    VLOG(V_DEBUG) << "allocating workspace";
  }

  float alpha = 1.f, beta = 0.f;
  checkCUDNNError(hipdnnConvolutionBackwardFilter(CudaCommon::cudnnHandle(),
                  &alpha, x_desc_, x.data<T>(),
                  y_desc_, dy.data<T>(),
                  conv_desc_, bwd_f_algo_, filter_workspace, filter_workspaceSizeInBytes_,
                  &beta, filter_desc_, df->mutable_data<T>()));
  checkCUDNNError(hipdnnConvolutionBackwardData(CudaCommon::cudnnHandle(),
                  &alpha, filter_desc_, filter.data<T>(),
                  y_desc_, dy.data<T>(),
                  conv_desc_, bwd_d_algo_, data_workspace, data_workspaceSizeInBytes_,
                  &beta, x_desc_, dx->mutable_data<T>()));
  checkCUDNNError(hipdnnConvolutionBackwardBias(CudaCommon::cudnnHandle(),
                  &alpha, y_desc_, dy.data<T>(),
                  &beta, bias_desc_, db->mutable_data<T>()));

  /*dy.DebugNumerical<T>();*/
  /*x.DebugNumerical<T>();*/
  /*filter.DebugNumerical<T>();*/
  /*df->DebugNumerical<T>();*/
  /*db->DebugNumerical<T>();*/
  /*dx->DebugNumerical<T>();*/
}

REGISTER_OP_IMPL_BUILDER(Key("Conv").Device("GPU"), ConvOpCudnn<float>);
REGISTER_OP_IMPL_BUILDER(Key(GetGradientName("Conv")).Device("GPU"), ConvOpCudnnGrad<float>);

} //namespace backend
