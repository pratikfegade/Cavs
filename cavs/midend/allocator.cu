#include "cavs/midend/allocator.h"
/*#include "cavs/midend/devices.h"*/
#include "cavs/util/macros_gpu.h"
#include "cavs/util/op_util.h"

namespace midend {

class GPUAllocator : public Allocator {
 public:
  GPUAllocator()
      : Allocator(DeviceTypeToString(GPU), GPU) {}
  void* AllocateRaw(size_t nbytes) override {
    VLOG(V_DEBUG) << "allocating " << nbytes << " bytes";
    void* ptr = NULL;
    checkCudaError(hipMalloc(&ptr, nbytes));
    checkCudaError(hipMemset(ptr, 0, nbytes));
    CHECK_NOTNULL(ptr);
#ifdef CORTEX_MEM_PROF
    Allocator::current_mem_usage += nbytes;
    if (Allocator::current_mem_usage > Allocator::max_mem_usage)
      Allocator::max_mem_usage.exchange(Allocator::current_mem_usage);
    Allocator::buf_size_map[ptr] = nbytes;
#endif
    return ptr;
  }
  void DeallocateRaw(void* buf) override {
#ifdef CORTEX_MEM_PROF
    Allocator::current_mem_usage -= Allocator::buf_size_map[buf];
    Allocator::buf_size_map.erase(buf);
#endif
    checkCudaError(hipFree(buf));
  }
  void InitWithZero(void* buf, size_t nbytes) override {
    checkCudaError(hipMemsetAsync(buf, 0, nbytes, hipStreamDefault));
  }
};

Allocator* gpu_allocator() {
  static GPUAllocator gpu_alloc;
  return &gpu_alloc;
}

REGISTER_STATIC_ALLOCATOR("GPU", gpu_allocator());

} //namespace midend
