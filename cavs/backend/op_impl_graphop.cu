#include "hip/hip_runtime.h"
#include "cavs/backend/op_impl.h"
#include "cavs/backend/functor_batched_memcpy.cuh"
#include "cavs/midend/graph_scheduler.h"
#include "cavs/midend/tensor.h"
#include "cavs/util/macros_gpu.h"
#include "cavs/util/op_util.h"

#include <iostream>
#include <string>

using ::midend::Tensor;
using ::midend::GraphSchedulerBase;
using std::vector;
using std::string;

namespace backend {

template <typename T>
class GraphGatherOp : public OpImpl {
 public:
  explicit GraphGatherOp(const OpDef& def) :
    OpImpl(def), count_(1), stream_(hipStreamDefault)  {

    CHECK(def.input_size()  == 0);
    CHECK(def.output_size() == 1);
    CHECK(def.shape_size()  == 1);
    for (auto d : def.shape(0).dim())
      count_ *= d;
    child_offset_ = GetSingleArg<int>(def, "Child");
    CHECK(child_offset_ >= 0);
  }

  void Compute(OpContext* context) override {
    //LOG(FATAL) << "Gather Operator needs further runtime support";
    Tensor* out = context->Output(0);
    GraphSchedulerBase* gs = context->graph_scheduler();
    const Tensor& inp = gs->GetMessagePasser(0);

    const vector<int>& gids = gs->GetJobId();
    context->SetDynDim(gids.size());
    context->ScaleOutputTensor();
    int stride = out->count()/out->dims(0);
    CHECK(stride == count_) << out->debug_info() << op_def_.DebugString();
    VLOG(V_DEBUG) << "Batching jobs of this round: " << gids.size();
    if (VLOG_IS_ON(V_DEBUG)) {
      string out;
      for (int id : gids) out += std::to_string(id) + "\t";
      VLOG(V_DEBUG) << out;
    }

    const vector<int>& tensor_ids_for_gather = gs->CurrentRoundTensorIdsForGather(child_offset_);
    if (VLOG_IS_ON(V_DEBUG)) {
      string out;
      for (int id : tensor_ids_for_gather) out += std::to_string(id) + "\t";
      VLOG(V_DEBUG) << out;
    }
    //for skewed batched trees, in the backward pass,
    //the root of one tree does not need to gather,
    //but the inode of other tree have to gather
    //so we loose this constraint
    /*CHECK(gids.size() == tensor_ids_for_gather.size() || tensor_ids_for_gather.empty());*/
    if (!stream_ && context->GetStreamID() != -1) {
      stream_ = StreamEventHandlePool::GetCudaStream(context->GetStreamID());
      VLOG(V_DEBUG) << "[Unary] Assign new stream with ID " << context->GetStreamID();
    }

    if (!tensor_ids_for_gather.empty()) {
      checkCudaError(hipMemcpyAsync(gs->gpu_idx_buf(), tensor_ids_for_gather.data(),
                     tensor_ids_for_gather.size()*sizeof(int), hipMemcpyHostToDevice, stream_));
      /*int blocksPerGrid = gids.size();*/
      int blocksPerGrid = tensor_ids_for_gather.size();
      /*int threadsPerBlock = stride;*/
      const int MAX_THREADS_IN_BLOCK = 1 << 10;
      int threadsPerBlock = (MAX_THREADS_IN_BLOCK > stride)? stride : MAX_THREADS_IN_BLOCK;
      BatchedDynamicSelectedInputSliceCopyKernel<T><<<blocksPerGrid, threadsPerBlock, 0, stream_>>>(
              out->mutable_data<T>(), stride, inp.data<T>(), stride, gs->gpu_idx_buf(), stride);
    }else {
      /*checkCudaError(hipMemset(out->mutable_data<T>(), 0, gids.size()*stride*sizeof(T)));*/
      int blocksPerGrid = gs->CurrentRoundTensorIdsForGatherInitialization().size();
      checkCudaError(hipMemcpyAsync(gs->gpu_idx_buf(), gs->CurrentRoundTensorIdsForGatherInitialization().data(),
                     blocksPerGrid*sizeof(int), hipMemcpyHostToDevice, stream_));
      const int MAX_THREADS_IN_BLOCK = 1 << 10;
      int threadsPerBlock = (MAX_THREADS_IN_BLOCK > stride)? stride : MAX_THREADS_IN_BLOCK;
      BatchedDynamicSelectedAssignZeroKernel<T><<<blocksPerGrid, threadsPerBlock, 0, stream_>>>(
              out->mutable_data<T>(), stride, gs->gpu_idx_buf(), stride);
    }
    checkCudaError(hipGetLastError());

    out->DebugNumerical<T>();

    // {
    //   std::cout << "[GATHERED] " << std::endl;
    //   vector<float> res(out->count());
    //   if (out->device_type() == GPU) {
    // 	checkCudaError(hipMemcpy(res.data(), out->data<float>(),
    // 				  out->count()*sizeof(float), hipMemcpyDeviceToHost));
    //   } else {
    // 	checkCudaError(hipMemcpy(res.data(), out->data<float>(),
    // 				  out->count()*sizeof(float), hipMemcpyHostToHost));
    //   }
    //   std::cout << "[PULL_OP] Gathered " << out->count() << std::endl;
    //   for (int i = 0; i < out->count(); ++i) {
    // 	std::cout << res[i] << " ";
    //   }
    //   std::cout << std::endl;
    // }
  }

 private:
  int count_;
  int child_offset_;
  hipStream_t stream_;
};

template <typename T>
class GraphScatterOp : public OpImpl {
 public:
  explicit GraphScatterOp(const OpDef& def) :
    OpImpl(def), stream_(hipStreamDefault) {

    child_offset_ = GetSingleArg<int>(def, "Child");
    CHECK(child_offset_ >= 0);
  }

  void Compute(OpContext* context) override {
    //LOG(FATAL) << "Scatter Operator needs further runtime support";
    const Tensor& inp = context->Input(0);
    Tensor* out = context->Output(0);
    CHECK(out->count() == inp.count())
          << "Input count:\t" << inp.count()
          << "\t" << inp.debug_size() << "Bytes\n"
          << "Output count:\t" << out->count()
          << "\t" << out->debug_size() << "Bytes";
    CHECK(inp.IsDynamicShape());
    CHECK(out->IsDynamicShape());
    CHECK(out->dims(0) == inp.dims(0));
    int stride = out->count()/out->dims(0);
    CHECK(stride == inp.count()/inp.dims(0));

    out->SetOffsetWithId(0);
    GraphSchedulerBase* gs = context->graph_scheduler();
    const vector<int>& gids = gs->GetJobId();
    VLOG(V_DEBUG) << "Batching jobs of this round: " << gids.size();
    if (VLOG_IS_ON(V_DEBUG)) {
      string out;
      for (int id : gids) out += std::to_string(id) + "\t";
      VLOG(V_DEBUG) << out;
    }

    const vector<int>& tensor_ids_for_scatter = gs->CurrentRoundTensorIdsForScatter(child_offset_);
    VLOG(V_DEBUG) << "tensor ids for scatter: " << tensor_ids_for_scatter.size();
    if (VLOG_IS_ON(V_DEBUG)) {
      string out;
      for (int id : tensor_ids_for_scatter) out += std::to_string(id) + "\t";
      VLOG(V_DEBUG) << out;
    }
    //for skewed batched trees, the root of one tree does not need to scatter,
    //but the inode of other tree have to scatter
    //so we loose this constraint
    /*CHECK(gids.size() == tensor_ids_for_scatter.size() || tensor_ids_for_scatter.empty());*/
    if (!stream_ && context->GetStreamID() != -1) {
      stream_ = StreamEventHandlePool::GetCudaStream(context->GetStreamID());
      VLOG(V_DEBUG) << "[Unary] Assign new stream with ID " << context->GetStreamID();
    }
    if (!tensor_ids_for_scatter.empty()) {
      checkCudaError(hipMemcpyAsync(gs->gpu_idx_buf(), tensor_ids_for_scatter.data(),
                     tensor_ids_for_scatter.size()*sizeof(int), hipMemcpyHostToDevice, stream_));
      int blocksPerGrid = tensor_ids_for_scatter.size();
      /*int threadsPerBlock = stride;*/
      const int MAX_THREADS_IN_BLOCK = 1 << 10;
      int threadsPerBlock = (MAX_THREADS_IN_BLOCK > stride)? stride : MAX_THREADS_IN_BLOCK;
      BatchedDynamicSelectedOutputSliceCopyKernel<T><<<blocksPerGrid, threadsPerBlock, 0, stream_>>>(
              out->mutable_data<T>(), stride, gs->gpu_idx_buf(), inp.data<T>(), stride, stride);
    }

    checkCudaError(hipGetLastError());
    out->DebugNumerical<T>();
  }

 private:
  int child_offset_;
  hipStream_t stream_;
};

template <typename T>
class GraphPushOp : public OpImpl {
 public:
  explicit GraphPushOp(const OpDef& def) :
    OpImpl(def), stream_(hipStreamDefault) {}
  void Compute(OpContext* context) override {
    //LOG(FATAL) << "Push Operator needs further runtime support";
    GraphSchedulerBase* gs = context->graph_scheduler();
    CHECK_NOTNULL(gs);
    const Tensor& inp = context->Input(0);
    Tensor* out = context->Output(0);
    //CHECK(out->count() >= inp.count())
    VLOG(V_DEBUG) << "Input count:\t" << inp.count()
                  << "\t" << inp.debug_size() << "Bytes\n"
                  << "Output count:\t" << out->count()
                  << "\t" << out->debug_size() << "Bytes";

    T* out_ptr = out->mutable_data<T>();
    CHECK(!out->IsFullShape());

    if (!stream_ && context->GetStreamID() != -1) {
      stream_ = StreamEventHandlePool::GetCudaStream(context->GetStreamID());
      VLOG(V_DEBUG) << "[Unary] Assign new stream with ID " << context->GetStreamID();
    }
    /*LOG(FATAL) << context->GetStreamID();*/
    checkCudaError(hipMemcpyAsync(out_ptr, inp.data<T>(),
                                   inp.count()*sizeof(T),
                                   hipMemcpyDeviceToDevice, stream_));
    /*ContinuousMemcpyKernel<<<BLOCKS_PER_GRID(inp.count()), THREADS_PER_BLOCK, 0, stream_>>>(*/
        /*out_ptr, inp.data<T>(), inp.count());*/
    gs->SetFuncRet(*out);

    inp.DebugNumerical<T>();
    out->DebugNumerical<T>();
  }

 private:
  hipStream_t stream_;
};

template <typename T>
class GraphPullOp : public OpImpl {
 public:
  explicit GraphPullOp(const OpDef& def) :
    OpImpl(def), stream_(hipStreamDefault)  {}

  void Compute(OpContext* context) override {
    //LOG(FATAL) << "Pull Operator needs further runtime support";
    GraphSchedulerBase* gs = context->graph_scheduler();
    CHECK_NOTNULL(gs);
    const Tensor& inp = gs->GetFuncArg();
    Tensor* out = context->Output(0);
    CHECK(inp.count() >= out->count())
          << "Input count:\t" << inp.count()
          << "\t" << inp.debug_size() << "Bytes\n"
          << "Output count:\t" << out->count()
          << "\t" << out->debug_size() << "Bytes";

    //out tensor must be local
    //if in tensor is a global tensor(in the backward of pull)
    //CHECK(inp.IsFullShape());
    const vector<int>& gids = gs->GetJobId();

    // {
    //   std::cout << "[PULL_OP] Pulling for gids " << std::endl;
    //   for (auto id: gids) {
    // 	std::cout << id << " " << std::endl;
    //   }
    //   std::cout << std::endl;
    //   std::cout << "Input count:\t" << inp.count()
    // 	     << "\t" << inp.debug_size() << "Bytes\n"
    // 	     << "Output count:\t" << out->count()
    // 	     << "\t" << out->debug_size() << "Bytes" << std::endl;
    // }




    context->SetDynDim(gids.size());
    context->ScaleOutputTensor();
    int stride = out->count()/out->dims(0);
    CHECK(out->dims(0) == gids.size());
    /*VLOG(V_DEBUG) << out->debug_info() << "\t" << out->debug_size();*/
    /*VLOG(V_DEBUG) << inp.debug_info() << "\t" << inp.debug_size();*/

    if (!stream_ && context->GetStreamID() != -1) {
      stream_ = StreamEventHandlePool::GetCudaStream(context->GetStreamID());
      VLOG(V_DEBUG) << "[Unary] Assign new stream with ID " << context->GetStreamID();
    }
    checkCudaError(hipMemcpyAsync(gs->gpu_idx_buf(), gids.data(),
                   gids.size()*sizeof(int), hipMemcpyHostToDevice, stream_));
    int blocksPerGrid = gids.size();
    /*int threadsPerBlock = stride;*/
    const int MAX_THREADS_IN_BLOCK = 1 << 10;
    int threadsPerBlock = (MAX_THREADS_IN_BLOCK > stride)? stride : MAX_THREADS_IN_BLOCK;
    checkCudaError(hipGetLastError());
    BatchedDynamicSelectedInputSliceCopyKernel<T><<<blocksPerGrid, threadsPerBlock, 0, stream_>>>(
            out->mutable_data<T>(), stride, inp.data<T>(), stride, gs->gpu_idx_buf(), stride);
    checkCudaError(hipGetLastError());

    // {
    //   vector<float> res(inp.count());
    //   if (inp.device_type() == GPU) {
    // 	checkCudaError(hipMemcpy(res.data(), inp.data<float>(),
    // 				  inp.count()*sizeof(float), hipMemcpyDeviceToHost));
    //   } else {
    // 	checkCudaError(hipMemcpy(res.data(), inp.data<float>(),
    // 				  inp.count()*sizeof(float), hipMemcpyHostToHost));
    //   }
    //   std::cout << "[PULL_OP] Pulled for gids " << std::endl;
    //   for (auto id: gids) {
    // 	std::cout << id << " " << res[id] << " " << std::endl;
    //   }
    // }

    inp.DebugNumerical<T>();
    out->DebugNumerical<T>();
  }

 private:
  hipStream_t stream_;
};

template <typename T>
class FunctionPushArgOp : public OpImpl {
 public:
  explicit FunctionPushArgOp(const OpDef& def) : OpImpl(def) {}
  void Compute(OpContext* context) override {
    //LOG(FATAL) << "here";
    const Tensor& inp = context->Input(0);
    GraphSchedulerBase* gs = context->graph_scheduler();
    CHECK_NOTNULL(gs);
    gs->SetFuncArg(inp);
    inp.DebugNumerical<T>();
  }
};

template <typename T>
class FunctionPopRetOp : public OpImpl {
 public:
  explicit FunctionPopRetOp(const OpDef& def) :
    OpImpl(def), stream_(hipStreamDefault) {}

  void Compute(OpContext* context) override {
    GraphSchedulerBase* gs = context->graph_scheduler();
    CHECK_NOTNULL(gs);
    const Tensor& inp = gs->GetFuncRet();
    Tensor* out = context->Output(0);
    VLOG(V_DEBUG) << inp.debug_info();
    VLOG(V_DEBUG) << out->debug_info();
    CHECK(inp.count() <= out->count())
      << inp.count() << "\t" << out->count();
    CHECK(inp.debug_size() >= out->debug_size())
        << inp.debug_size() << "\t" << out->debug_size();
    VLOG(V_DEBUG) << inp.debug_info();
    VLOG(V_DEBUG) << out->debug_info();

    CHECK(inp.IsDynamicShape());
    //for the backward, the gradient of lower layer output may not be dynamic
    //for example, the placeholder of layer0
    /*CHECK(out->IsDynamicShape());*/
    int stride = inp.count()/inp.dims(0);
    int out_dyn_dim = out->dims(0);
    //for the backward, the shape of lower layer output is arbitrary
    //for example, the placeholder may be {2, 4} (batch, time_step)
    //here, the inp shape may be {1, 1} (serial model) or {2, 1} (batch mode)
    /*CHECK(stride == out->count()/out_dyn_dim);*/
    const vector<int>& tids2gids= gs->TensorIdsToJobIds();
    for (int i = 0; i < tids2gids.size(); i++) {
      VLOG(V_DEBUG) << "i: " << i << "\tgid: " << tids2gids[i];
    }
    checkCudaError(hipMemcpyAsync(gs->gpu_idx_buf(), tids2gids.data(),
                   tids2gids.size()*sizeof(int), hipMemcpyHostToDevice, stream_));
    int blocksPerGrid = tids2gids.size();
    /*int threadsPerBlock = stride;*/
    const int MAX_THREADS_IN_BLOCK = 1 << 10;
    VLOG(V_DEBUG) << blocksPerGrid;
    VLOG(V_DEBUG) << stride;
    checkCudaError(hipGetLastError());
    int threadsPerBlock = (MAX_THREADS_IN_BLOCK > stride)? stride : MAX_THREADS_IN_BLOCK;
    BatchedDynamicSelectedOutputSliceCopyKernel<T><<<blocksPerGrid, threadsPerBlock, 0, stream_>>>(
            out->mutable_data<T>(), stride, gs->gpu_idx_buf(), inp.data<T>(), stride, stride);
    checkCudaError(hipGetLastError());

    inp.DebugNumerical<T>();
    out->DebugNumerical<T>();
  }

 private:
  hipStream_t stream_;
};

REGISTER_OP_IMPL_BUILDER(Key("Pull").Device("GPU"),    GraphPullOp<float>);
REGISTER_OP_IMPL_BUILDER(Key("Push").Device("GPU"),    GraphPushOp<float>);
REGISTER_OP_IMPL_BUILDER(Key("Scatter").Device("GPU"), GraphScatterOp<float>);
REGISTER_OP_IMPL_BUILDER(Key("Gather").Device("GPU"),  GraphGatherOp<float>);
REGISTER_OP_IMPL_BUILDER(Key("FunctionPushArg").Device("GPU"), FunctionPushArgOp<float>);
REGISTER_OP_IMPL_BUILDER(Key("FunctionPopRet").Device("GPU"), FunctionPopRetOp<float>);

} //namespace backend
